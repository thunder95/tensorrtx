#include "hip/hip_runtime.h"
#include "yolov5_face_plugin.h"
#include "stdio.h"
#include <iostream>
#include <cassert>
#include <memory>
#include<math.h>

#ifndef CUDA_CHECK

#define CUDA_CHECK(callstr)                                                                    \
    {                                                                                          \
        hipError_t error_code = callstr;                                                      \
        if (error_code != hipSuccess) {                                                       \
            std::cerr << "CUDA error " << error_code << " at " << __FILE__ << ":" << __LINE__; \
            assert(0);                                                                         \
        }                                                                                      \
    }

#endif

namespace nvinfer1
{
    yolov5FacePlugin::yolov5FacePlugin()
    {
        conf_thresh = yolov5FaceConfig::CONF_THRESH;
        refer_rows_1 = 3 * yolov5FaceConfig::INPUT_H * yolov5FaceConfig::INPUT_W / (8.0 * 8.0);
        refer_rows_2 = refer_rows_1 + 3 * yolov5FaceConfig::INPUT_H * yolov5FaceConfig::INPUT_W / (16.0 * 16.0);
        refer_rows_3 = refer_rows_2 + 3 * yolov5FaceConfig::INPUT_H * yolov5FaceConfig::INPUT_W / (32.0 * 32.0);
        std::cout<<"init decode plugin" <<std::endl;
    }

    yolov5FacePlugin::~yolov5FacePlugin()
    {
        std::cout<<"destroy yolov5_face plugin"<<std::endl;
    }

    // create the plugin at runtime from a byte stream
    yolov5FacePlugin::yolov5FacePlugin(const void* data, size_t length)
    {
    }

    void yolov5FacePlugin::serialize(void* buffer) const
    {
    }

    size_t yolov5FacePlugin::getSerializationSize() const
    {
        return 0;
    }

    int yolov5FacePlugin::initialize()
    {
        return 0;
    }

    Dims yolov5FacePlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
    {
        //output the result to channel
        int totalsize = yolov5FaceConfig::MAX_OUT * sizeof(yolov5FaceConfig::FaceBox) / sizeof(float);
        return Dims3(totalsize + 1, 1, 1);

    }

    // Set plugin namespace
    void yolov5FacePlugin::setPluginNamespace(const char* pluginNamespace)
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* yolov5FacePlugin::getPluginNamespace() const
    {
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType yolov5FacePlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
    {
        return DataType::kFLOAT;
    }

    // Return true if output tensor is broadcast across a batch.
    bool yolov5FacePlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const
    {
        return false;
    }

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool yolov5FacePlugin::canBroadcastInputAcrossBatch(int inputIndex) const
    {
        return false;
    }

    void yolov5FacePlugin::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput)
    {
    }

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void yolov5FacePlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator)
    {
    }

    // Detach the plugin object from its execution context.
    void yolov5FacePlugin::detachFromContext() {}

    const char* yolov5FacePlugin::getPluginType() const
    {
        return "YOLOV5FACE_TRT";
    }

    const char* yolov5FacePlugin::getPluginVersion() const
    {
        return "1";
    }

    void yolov5FacePlugin::destroy()
    {
        delete this;
    }

    // Clone the plugin
    IPluginV2IOExt* yolov5FacePlugin::clone() const
    {
        yolov5FacePlugin *p = new yolov5FacePlugin();
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }

    __device__ float Logist(float data){ return 1./(1. + expf(-data)); }; //sigmoid func
    __device__ float dev_anchors_grid[3][6] = {4,5,  8,10,  13,16, 23,29,  43,55,  73,105, 146,217,  231,300,  335,433};
    __device__ int dev_strides[3] = {8, 16, 32};

    //计算, 输入大小 18900 * 16
    __global__ void CalDetection(const float *input, float *output, int refer_rows_1, int refer_rows_2,
            int refer_rows_3, int num_elem, int output_elem) {
            int idx = threadIdx.x + blockDim.x * blockIdx.x;
            if (idx >= num_elem) return;

            const float* curInput = input + idx * 16; //每行第一个, 16暂时写死
            int bn_idx = idx / refer_rows_3;      //batch_id, total_grid=18900
            idx %= refer_rows_3;                  //offset idx in batch_id

            // 过滤置信度
            float cur_conf = Logist(*(curInput + 4));
            if (cur_conf <= yolov5FaceConfig::CONF_THRESH) //0.3
                return;
            //printf("ori_val: %f, conf: %f, thresh: %f\n", *(curInput + 4), cur_conf, yolov5FaceConfig::CONF_THRESH);

            float *res_count = output + bn_idx * output_elem;
            int count = (int)atomicAdd(res_count, 1); //总数累加
            if (count >= yolov5FaceConfig::MAX_OUT) return;

            //判断位于哪个层
            int layer_idx = 0;
            if (idx > refer_rows_2) {
                layer_idx = 2;
                idx -= refer_rows_2;
            } else if (idx > refer_rows_1) {
                layer_idx = 1;
                idx -= refer_rows_1;
            }

            int cur_stride = dev_strides[layer_idx];
            int cur_w = yolov5FaceConfig::INPUT_W / cur_stride;
            int cur_h = yolov5FaceConfig::INPUT_H / cur_stride;
            int h_idx_all = idx / cur_w; //h方向堆叠的全局idx
            int h_idx = h_idx_all % cur_h; //grid范围内的索引
            int anchor_idx = (h_idx_all / cur_h) % 3; //考虑归属哪一个anchor
            int w_idx = idx % cur_w; //横向对应的索引

            //构造检测框
            char* data = (char *)res_count + sizeof(float) + count * sizeof(yolov5FaceConfig::FaceBox);
            yolov5FaceConfig::FaceBox* det = (yolov5FaceConfig::FaceBox*)(data);

            //xywhs c
            float bw = pow((Logist(*(curInput + 2)) * 2), 2) * dev_anchors_grid[layer_idx][2 * anchor_idx]; //w
            float bh = pow((Logist(*(curInput + 3)) * 2), 2) * dev_anchors_grid[layer_idx][2 * anchor_idx + 1]; //h
            det->bbox[0] = (Logist(*(curInput)) * 2. - 0.5 + w_idx) * cur_stride - bw / 2.0; //x1
            det->bbox[1] = (Logist(*(curInput + 1)) * 2. - 0.5 + h_idx) * cur_stride - bh / 2.0; //y1
            det->bbox[2] = det->bbox[0] + bw; //x2
            det->bbox[3] = det->bbox[1] + bh; //y2
            det->bbox[4] = cur_conf * Logist(*(curInput+ 15)); //score
            det->conf = cur_conf;

            //landmarks x1 y1 -> x5 y5
            det->landmarks[0] = (*(curInput+ 5)) * dev_anchors_grid[layer_idx][2 * anchor_idx] + w_idx * cur_stride;
            det->landmarks[1] = (*(curInput+ 6)) * dev_anchors_grid[layer_idx][2 * anchor_idx + 1] + h_idx * cur_stride;
            det->landmarks[2] = (*(curInput+ 7)) * dev_anchors_grid[layer_idx][2 * anchor_idx] + w_idx * cur_stride;
            det->landmarks[3] = (*(curInput+ 8)) * dev_anchors_grid[layer_idx][2 * anchor_idx + 1] + h_idx * cur_stride;
            det->landmarks[4] = (*(curInput+ 9)) * dev_anchors_grid[layer_idx][2 * anchor_idx] + w_idx * cur_stride;
            det->landmarks[5] = (*(curInput+ 10)) * dev_anchors_grid[layer_idx][2 * anchor_idx + 1] + h_idx * cur_stride;
            det->landmarks[6] = (*(curInput+ 11)) * dev_anchors_grid[layer_idx][2 * anchor_idx] + w_idx * cur_stride;
            det->landmarks[7] = (*(curInput+ 12)) * dev_anchors_grid[layer_idx][2 * anchor_idx + 1] + h_idx * cur_stride;
            det->landmarks[8] = (*(curInput+ 13)) * dev_anchors_grid[layer_idx][2 * anchor_idx] + w_idx * cur_stride;
            det->landmarks[9] = (*(curInput+ 14)) * dev_anchors_grid[layer_idx][2 * anchor_idx + 1] + h_idx * cur_stride;
        }

    void yolov5FacePlugin::forwardGpu(const float *const * inputs, float * output, hipStream_t stream, int batchSize)
    {

        int outputElem = 1 + yolov5FaceConfig::MAX_OUT * sizeof(yolov5FaceConfig::FaceBox) / sizeof(float);

        for (int idx = 0; idx < batchSize; ++idx) {
            CUDA_CHECK(hipMemset(output + idx * outputElem, 0, sizeof(float))); //set total_num=0
        }

        int total_num_elem = refer_rows_3 * batchSize;
        //std::cout<<"total_num_elem: "<<total_num_elem << "row num: "<<row_num<<" batchsize:" << batchSize <<std::endl;

        CalDetection << < (total_num_elem + thread_count_ - 1) / thread_count_, thread_count_ , 0, stream >> > (inputs[0],
                output, refer_rows_1, refer_rows_2, refer_rows_3, total_num_elem, outputElem);
    }

    int yolov5FacePlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
    {
        //GPU
        //CUDA_CHECK(hipStreamSynchronize(stream));
        forwardGpu((const float *const *)inputs, (float *)outputs[0], stream, batchSize);
        return 0;
    };

    PluginFieldCollection yolov5FacePluginCreator::mFC{};
    std::vector<PluginField> yolov5FacePluginCreator::mPluginAttributes;

    yolov5FacePluginCreator::yolov5FacePluginCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* yolov5FacePluginCreator::getPluginName() const
    {
        return "YOLOV5FACE_TRT";
    }

    const char* yolov5FacePluginCreator::getPluginVersion() const
    {
        return "1";
    }

    const PluginFieldCollection* yolov5FacePluginCreator::getFieldNames()
    {
        return &mFC;
    }

    IPluginV2IOExt* yolov5FacePluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
    {
        yolov5FacePlugin* obj = new yolov5FacePlugin();
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt* yolov5FacePluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
    {
        // This object will be deleted when the network is destroyed, which will
        // call PReluPlugin::destroy()
        yolov5FacePlugin* obj = new yolov5FacePlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

}

//图像预处理
__global__ void yolov5FacePreprocessKernel(const unsigned char*src,int srcWidth,int srcHeight,int srcPitch,float *dst,
    int dstWidth,int dstHeight, int write_x, int write_y, float resize_w, float resize_h)
{
    double srcXf;
    double srcYf;
    int srcX;
    int srcY;
    double u;
    double v;
    int dstOffset;

    int y = blockIdx.y*blockDim.y+threadIdx.y;
    int x = blockIdx.x*blockDim.x+threadIdx.x;

    if(x>=dstWidth || y>=dstHeight)
        return;

    //int write_x,write_y;
    //float resize_w,resize_h;
    double r_w = dstWidth / (srcWidth*1.0);
    double r_h = dstHeight / (srcHeight*1.0);
    if (r_h > r_w) {
        resize_w = dstWidth;
        resize_h = r_w * srcHeight;
        write_x = 0;
        write_y = (dstHeight - resize_h) / 2;
    } else {
        resize_w = r_h * srcWidth;
        resize_h = dstHeight;
        write_x = (dstWidth - resize_w) / 2;
        write_y = 0;
    }

    if((x >= write_x) && (x < write_x + resize_w) && (y >= write_y) && (y < write_y + resize_h))
    {
        srcXf=  (x - write_x) * ((float)(srcWidth/resize_w)) ;
        srcYf =  (y - write_y) * ((float)(srcHeight/resize_h));
        srcX = (int)srcXf;
        srcY = (int)srcYf;
        u= srcXf - srcX;
        v = srcYf - srcY;

        //r chanel
        if(y*dstWidth+x >= dstWidth*dstHeight)
        {
            return;
        }

        if(srcY*srcPitch+srcX >= srcPitch*srcHeight ||
           (srcY+1)*srcPitch+srcX >= srcPitch*srcHeight ||
           srcY*srcPitch+(srcX+1) >= srcPitch*srcHeight ||
           (srcY+1)*srcPitch+(srcX+1) >= srcPitch*srcHeight)
        {
            return;
        }

        dstOffset =(y*dstWidth+x) + 2 * dstWidth * dstHeight;
        if(dstOffset >= dstWidth * dstHeight * 3)
        {
            return;
        }

        dst[dstOffset] = 0;
        dst[dstOffset]+=(1-u)*(1-v)*src[srcY*srcPitch+srcX*3];
        dst[dstOffset]+=(1-u)*v*src[(srcY+1)*srcPitch+srcX*3];
        dst[dstOffset]+=u*(1-v)*src[srcY*srcPitch+(srcX+1)*3];
        dst[dstOffset]+= u*v*src[(srcY+1)*srcPitch+(srcX+1)*3];
        dst[dstOffset] = dst[dstOffset] / 255.0;

        //g chanel
        dstOffset =(y*dstWidth+x) + dstWidth * dstHeight;
        if(dstOffset >= dstWidth * dstHeight * 3)
        {
            return;
        }

        dst[dstOffset] = 0;
        dst[dstOffset]+=(1-u)*(1-v)*src[srcY*srcPitch+srcX*3+1];
        dst[dstOffset]+=(1-u)*v*src[(srcY+1)*srcPitch+srcX*3+1];
        dst[dstOffset]+=u*(1-v)*src[srcY*srcPitch+(srcX+1)*3+1];
        dst[dstOffset]+= u*v*src[(srcY+1)*srcPitch+(srcX+1)*3+1];
        dst[dstOffset] = dst[dstOffset] / 255.0;

        //b chanel
        dstOffset =(y*dstWidth+x) ;
        if(dstOffset >= dstWidth * dstHeight * 3)
        {
            return;
        }

        dst[dstOffset] = 0;
        dst[dstOffset]+=(1-u)*(1-v)*src[srcY*srcPitch+srcX*3+2];
        dst[dstOffset]+=(1-u)*v*src[(srcY+1)*srcPitch+srcX*3+2];
        dst[dstOffset]+=u*(1-v)*src[srcY*srcPitch+(srcX+1)*3+2];
        dst[dstOffset]+= u*v*src[(srcY+1)*srcPitch+(srcX+1)*3+2];
        dst[dstOffset] = dst[dstOffset] / 255.0;

    } else
    {
        if(y*dstWidth+x >= dstWidth*dstHeight)
        {
            return;
        }

        //r chanel
        int dstOffset =(y*dstWidth+x) + 2 * dstWidth * dstHeight;
        if(dstOffset >= dstWidth * dstHeight * 3)
        {
            return;
        }

        dst[dstOffset] = 128;
        dst[dstOffset] = dst[dstOffset] / 255.0;

        //g chanel
        dstOffset =(y*dstWidth+x) + dstWidth * dstHeight;
        if(dstOffset >= dstWidth * dstHeight * 3)
        {
            return;
        }

        dst[dstOffset] = 128;
        dst[dstOffset] = dst[dstOffset] / 255.0;

        //b chanel
        dstOffset =(y*dstWidth+x);
        if(dstOffset >= dstWidth * dstHeight * 3)
        {
            return;
        }

        dst[dstOffset] = 128;
        dst[dstOffset] = dst[dstOffset] / 255.0;
    }
}

void yolov5FacePreprocess(const unsigned char*src,int srcWidth,int srcHeight,int srcPitch, float* dst,int dstWidth,
    int dstHeight, int write_x, int write_y, float resize_w, float resize_h, hipStream_t stream)
{
    int uint = 16;
    dim3 grid((dstWidth+uint-1)/uint,(dstHeight+uint-1)/uint);
    dim3 block(uint,uint);
    yolov5FacePreprocessKernel<<<grid,block,0,stream>>>(src, srcWidth, srcHeight,srcPitch,dst, dstWidth, dstHeight,
        write_x, write_y, resize_w, resize_h);
}
